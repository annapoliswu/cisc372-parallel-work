
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
//#include <sys/time.h>

//#include <opencv2/opencv.hpp>

__global__ void mykernel(int *arr, int *stop){
	int id = blockIdx.x * blockDim.x + threadIdx.x;  //unique global id of thread
	int numThreads = gridDim.x * blockDim.x;  //total num threads in grid in x direction
	int localsum = 0;
	
	for (int i = id; i < *stop; i+= numThreads){
		double tmp = sin(i*1.0);
		double tmp2 = tmp*tmp;
		int z = (int)(tmp2*10000.0);
		localsum = (localsum + z) % 10000;
	}
	printf(" %d ", localsum);
	
	arr[id] = localsum;
	/*
	if(id < *stop){
		double tmp = sin(id*1.0);
		double tmp2 = tmp*tmp;
		int z = (int)(tmp2*10000.0);
		arr[id] =  z % 10000;
	}
	*/
}


int main(int argc, char *argv[]){
	//assert(argc==2);
	int stop = (int)atol(argv[1]);
	assert(stop >= 1.0);

	printf("Hello World!\n");
	int blocks = 4;
	int threads = 5;
	int result = 0;

	int *arr;
	int arrsize; 
	if(blocks*threads < stop){
		arrsize = blocks*threads;
	}else{
		arrsize = stop;
	}
	arr = (int *)malloc(sizeof(int)*arrsize); //memory in cpu 
	
	int *devarr;
	int *devstop;
	hipMalloc((int**) &devarr , sizeof(int)*arrsize); //mem in gpu
	hipMalloc((int**) &devstop , sizeof(int)); //mem in gpu

	hipMemcpy(devarr, arr, sizeof(int)*arrsize, hipMemcpyHostToDevice); //transfer
	hipMemcpy(devstop, &stop, sizeof(int), hipMemcpyHostToDevice); //transfer
	mykernel<<<blocks,threads>>>(devarr, devstop);	//1,1 block, threads- launch config
	hipMemcpy(arr, devarr, sizeof(int)*arrsize, hipMemcpyDeviceToHost);
	
	printf("arrsize: %d\n", arrsize);
	
	for(int i = 0; i<arrsize; i++){
		//printf(" %d ", arr[i]);
		result = (result + arr[i]) % 10000;
	}
	
	printf("PIN is: %d\n",result);
	//scanf("%d%d", a, b);


	free(arr);
	hipFree(devarr);
	hipFree(devstop);

	return 0;
}